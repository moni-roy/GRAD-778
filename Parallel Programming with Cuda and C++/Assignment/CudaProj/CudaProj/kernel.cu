﻿
#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t calcCubesCuda(int *c, const int *a, unsigned int size);
void initializeArray(int* a, unsigned int size);
void calcCubesCPU(int* c, const int* a, unsigned int size);

__global__ void calcCubes(int* out, const int* in) {
    int index = blockIdx.x; // Onedimensional blocks of GPU threads, each block with 1 thread
    out[index] = in[index] * in[index] * in[index];
}


int main()
{
    const int arraySize = 500000;
    int* a = new int[arraySize];

    initializeArray(a, arraySize);

    int* c = new int[arraySize];

    //calcCubesCPU(c, a, arraySize);

    // Add vectors in parallel.
    hipError_t cudaStatus = calcCubesCuda(c, a, arraySize);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    printf("a^3 = {");
    for (int i = 0; i < 100; i++) {
        if (i) printf(", ");
        printf("%d", c[i]);
    }
    printf("}\n\n");

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t calcCubesCuda(int *c, const int *a, unsigned int size)
{
    int *dev_a = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    calcCubes <<<size, 1 >>> (dev_c, dev_a);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    
    return cudaStatus;
}

void initializeArray(int* a, unsigned int size)
{
    for (int i = 0; i < size; i++) {
        a[i] = i % 10;
    }
}

void calcCubesCPU(int* c, const int* a, unsigned int size)
{
    for (int i = 0; i < size; i++) {
        c[i] = a[i] * a[i] * a[i]; // calculate cubes of ai
    }
}
